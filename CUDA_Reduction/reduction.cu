/******************************************************
 * CUDA Sum Reduction
 * By: Sairam Krishnan
 * Date: May 6, 2014
 * Compile command: nvcc -arch=sm_20 reduction.cu
 ******************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10
#define NTHRDS 4
#define NBLKS (((N) + (NTHRDS-1)) / (NTHRDS))

__global__ void sumReduction(int *input, int *output) {
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   int sum = 0, i;
   __shared__ int temp[NTHRDS];

   //Load the values into shared memory.
   temp[threadIdx.x] = input[index];
   //Wait for all threads in the current block to finish loading values.
   __syncthreads();
 
   //Offload the reduction work for this block to thread 0.
   if (threadIdx.x != 0)
      return;
   for (i = 0; i<blockDim.x; i++) { 
      if (index+i >= N) 
	break;
      sum += temp[i];
   }

   //Atomic add to prevent inteference from threads outside this block
   atomicAdd(output, sum);
}

int main() {
   int input[N], output, i;
   int *devInput, *devOutput;

   for (i = 0; i<N; i++)
	input[i] = i+1;
   
   hipMalloc(&devInput, N*sizeof(int));
   hipMalloc(&devOutput, sizeof(int));
   hipMemcpy(devInput, input, N*sizeof(int), hipMemcpyHostToDevice);
   hipMemset(devOutput, 0, sizeof(int));

   sumReduction <<<NBLKS, NTHRDS>>>(devInput, devOutput);

   hipMemcpy(&output, devOutput, sizeof(int), hipMemcpyDeviceToHost);
   
   printf("%d\n", output);

   hipFree(devOutput);
   hipFree(devInput);
   return 0;
}
