
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
   *c = (*a) + (*b);
}

__global__ void multiply(int *a, int *b, int *c) {
   *c = (*a) * (*b);
}

__global__ void subtract(int *a, int *b, int *c) {
   *c = (*a) - (*b);
}

__global__ void divide(int *a, int *b, int *c) {
   *c = (*a) / (*b);
}

int main() {
  int host1, host2, output; //host variables
  char op; //host variable
  int *device1, *device2, *device3; //device memory

  //Allocate memory for device vars.
  hipMalloc((void **)&device1, sizeof(int));
  hipMalloc((void **)&device2, sizeof(int));
  hipMalloc((void **)&device3, sizeof(int));

  //Read 2 integers. Store values in host variables.
  printf("Enter two integers: ");
  fscanf(stdin, "%d %d %c", &host1, &host2, &op);
  
  //Transfer values from host to device.
  hipMemcpy(device1, &host1, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device2, &host2, sizeof(int), hipMemcpyHostToDevice);

  //Launch add kernel on GPU with given parameters.
  switch(op) {
     case '+': add <<< 1,1 >>>(device1, device2, device3); break;
     case '*': multiply <<< 1,1 >>>(device1, device2, device3); break;
     case '-': subtract <<< 1,1 >>>(device1, device2, device3); break;
     case '/': divide <<< 1,1 >>>(device1, device2, device3); break;
  }
  //Get result from device to host.
  hipMemcpy(&output, device3, sizeof(int), hipMemcpyDeviceToHost);

  //Print result.
  printf("%d %c %d = %d\n", host1, op, host2, output);

  //Free all variables.
  hipFree(device1);
  hipFree(device2);
  hipFree(device3);

  return 0;
}
